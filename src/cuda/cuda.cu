#include "hip/hip_runtime.h"
/**
 * Posix compliant version of CSA-Rainbow-Table-Tool
 * 
 * Copyright (C) 2015 NEngine Team
 * 
 * This file is part of CSA-Rainbow-Table-Tool.
 * 
 * CSA-Rainbow-Table-Tool is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * CSA-Rainbow-Table-Tool is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with CSA-Rainbow-Table-Tool.  If not, see <http://www.gnu.org/licenses/>.
 * 
 * This software is based on the windows version of CSA-Rainbow-Table-Tool
 * 
 * Copyright (C) 2012  Colibri <colibri.dvb@googlemail.com>  
 * http://colibri.net63.net/ alias http://colibri-dvb.info/
 * 
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <> 
#include <assert.h>
//#include "cuda_dvbcsa.h"
//#include "cuda_dvbcsa_pv.h"
#include "hip/hip_runtime.h"

#include "../shared/define.h"
#include "../shared/table.h"

/*#ifdef COPY_TO_SHARED_csa_stream_cdef
__shared__ uint16_t csa_stream_cdef[0x400];
__constant__ const uint16_t const_csa_stream_cdef[0x400] = 
#else
__constant__ const uint16_t csa_stream_cdef[0x400] = 
#endif
TABLE_CSA_STREAM_CDEF*/

/*#ifdef COPY_TO_SHARED_sbox
__shared__ uint16_t sbox[7][32];
__constant__ const uint16_t const_sbox[7][32] = 
#else
__constant__ const uint16_t sbox[7][32] = 
#endif
TABLE_SBOX*/

//#ifdef COPY_TO_SHARED_csa_stream_out
//__shared__  uint8_t csa_stream_out[16];
//__constant__  const uint8_t const_csa_stream_out[16] = 
//#else
//__constant__  const uint8_t csa_stream_out[16] = 
//#endif
//TABLE_CSA_STREAM_OUT

//#ifdef COPY_TO_SHARED_csa_key_perm
//__device__ __shared__ static uint8_t csa_key_perm[64];
//__device__ __constant__ static const uint8_t const_csa_key_perm[64] = 
//#else
//__device__ __constant__ static const uint8_t csa_key_perm[64] = 
//#endif
//TABLE_CSA_KEY_PERM

#ifdef COPY_TO_SHARED_csa_block_perm
__device__	__shared__ static uint8_t csa_block_perm[256];
__device__	__constant__ static const uint8_t const_csa_block_perm[256] = 
#else
__device__	__constant__ static const uint8_t csa_block_perm[256] = 
#endif
TABLE_CSA_BLOCK_PERM

#ifdef COPY_TO_SHARED_block_sbox
__device__ __shared__ uint8_t block_sbox[256];
__device__ __constant__ const uint8_t const_block_sbox[256] = 
#else
__device__ __constant__ const uint8_t block_sbox[256] = 
#endif
TABLE_BLOCK_SBOX

// CUDA kernel to pause for at least num_cycle cycles
__global__ void sleep(int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles) {
        cycles = clock64() - start;
    }
}

// Returns number of cycles required for requested seconds
static int64_t get_cycles(float seconds)
{
    // Get device frequency in KHz
    int64_t Hz;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    Hz = int64_t(prop.clockRate) * 1000;

    // Calculate number of cycles to wait
    int64_t num_cycles;
    num_cycles = (int64_t)(seconds * Hz);
  
    return num_cycles;
}

/*__device__ static uint8_t csa_stream_rotate(register uint32_t pqzyx,
	register uint32_t x)
csa_stream_rotate_code_block*/

/*__device__ static uint32_t csa_stream_sboxes(register uint64_t A)
csa_stream_sboxes_code_block*/

/*__device__ static uint32_t csa_stream_B_sel(register uint64_t B)
csa_stream_B_sel_code_block*/

/*__device__ static p_inline uint32_t csa_stream_cfed(uint32_t pqzyx,
	uint32_t cfed)
csa_stream_cfed_code_block*/

/*__device__ static void csa_stream_round(uint64_t *A,uint64_t* B,
	uint32_t* pqzyx, uint32_t *cfed)
csa_stream_round_code_block*/

__device__ void KeySchedule07_Xor(uchar *ksfull, const uchar *Cw8)
KeySchedule07_Xor_code_block

__device__ void block_encrypt_dataOnly_ulong(const dvbcsa_keys_t key,
	ulong *data)
block_encrypt_dataOnly_ulong_code_block


/** variante 1 */
__global__ void kernel_csa_rainbow_do_x_rounds(uchar *Cw6Array_d,
	ulong *CurRoundNrArray_d, ulong *RemRoundsArray_d, DEBUG_STRUCT *Debug_d)
{
	/**
	 * Get Cw6Array for single crypt value to cmp with rainbow table end values
	 * if _all_ values in RemRoundsArray_d is 0 then Cw6Array_d has the final
	 * value else the kernel must called again
	 */
	uchar Cw[8];
	//ulong dwCw;
	int Offset;
	Offset = (blockIdx.x * THREADS_RB) + threadIdx.x;

	if((BLOCKS_RB * THREADS_RB) != RAINBOW_CHAIN_LEN)
	{
		return;//error
	}

	// optimize by copy const to shared (for all treads)
	// till sync treads

	COPY_CONST_TO_SHARED

	/** sync */
	__syncthreads();

	uchar *ParCw6;
	ParCw6 = &Cw6Array_d[Offset * 6];
	ulong *ParCurRoundNr;
	ParCurRoundNr = &CurRoundNrArray_d[Offset];
	ulong *ParRemRounds;
	ParRemRounds = &RemRoundsArray_d[Offset];

	int i;
//	int j;
	int m;
//	int k;
	int RoundNr;

	ulong data[8];

#if 1

	for(m=0; m<RAINBOW_CHAIN_LEN_PER_KERNEL; m++)
	{
		RoundNr = *ParCurRoundNr;
		if(*ParRemRounds <= 0)
		{
			break;
		}

		//copy
		for(i=0; i<8; i++)
		{
			data[i] = ParCw6[i];
		}

		//checksum
		Cw[0] = data[0];
		Cw[1] = data[1];
		Cw[2] = data[2];
		Cw[3] = data[0] + data[1] + data[2];
		Cw[4] = data[3];
		Cw[5] = data[4];
		Cw[6] = data[5];
		Cw[7] = data[3] + data[4] + data[5];

		{
			uchar ksfull[7*8];
			KeySchedule07_Xor(ksfull, Cw);

			//plain input
			for(i=0; i<8; i++)
			{
				data[i] = 0;
			}

			//Enc
			for(i=0; i<23; i++)
			{
				block_encrypt_dataOnly_ulong(ksfull, data);
			}
		}

		//Reduction part 2 (XOR RoundNr)
		data[2] ^= ((RoundNr>>24)&0xFF);
		data[3] ^= ((RoundNr>>16)&0xFF);
		data[4] ^= ((RoundNr>>8)&0xFF);
		data[5] ^= (RoundNr&0xFF);


		*ParCurRoundNr = *ParCurRoundNr + 1;
		*ParRemRounds = *ParRemRounds - 1;

		//copy cw end
		for(i=0; i<6; i++)
		{
			ParCw6[i] = data[i];
		}

	}
#endif
}

/**  variante 1 */
__global__ void kernel_csa_rainbow_find_a(uchar *Cw6EndArray_d,
	ulong *RoundNrArray_d, uchar *CryptArray_d, DEBUG_STRUCT *Debug_d)
{
	/**
	 * Get Cw6Array for single crypt value to cmp with rainbow table end values
	 * if _all_ values in RoundNrArray is set to the last round nr then
	 * Cw6Array_d contains the Cw6 _end_ values
	 * else the kernel must called again
	 * input (first call):		RoundNrArray_d, CryptArray_d
	 * input (other calls):		Cw6EndArray_d,	RoundNrArray_d, CryptArray_d
	 * output (not last call):	Cw6EndArray_d,	RoundNrArray_d, CryptArray_d
	 * output (last call):		Cw6EndArray_d
	 */
	uchar Cw[8];
	//ulong dwCw;
	int Offset;
	Offset = (blockIdx.x * THREADS_RB) + threadIdx.x;

	if((BLOCKS_RB * THREADS_RB) != RAINBOW_CHAIN_LEN)
	{
		return;//error
	}

	// optimize by copy const to shared (for all treads)
	// till sync treads

	COPY_CONST_TO_SHARED

	/** sync */
	__syncthreads();

	uchar *ParCrypt;
	ParCrypt = &CryptArray_d[Offset * 8];
	ulong *ParRoundNr;
	ParRoundNr = &RoundNrArray_d[Offset];
	uchar *ParCw6End;
	ParCw6End = &Cw6EndArray_d[Offset * 6];

	int i;
//	int k;
//	int j;
	int m;
	int RoundNr;

	ulong data[8];

	for(m=0; m<RAINBOW_CHAIN_LEN_PER_KERNEL; m++)
	{
		RoundNr = *ParRoundNr;
		if(RoundNr >= (BLOCKS_RB * THREADS_RB))
		{
			break;
		}

		//copy crypt
		for(i=0; i<8; i++)
		{
			data[i] = ParCrypt[i];
		}

		//Reduction part 2 (XOR RoundNr)
		data[2] ^= ((RoundNr>>24)&0xFF);
		data[3] ^= ((RoundNr>>16)&0xFF);
		data[4] ^= ((RoundNr>>8)&0xFF);
		data[5] ^= (RoundNr&0xFF);

		//checksum
		Cw[0] = data[0];
		Cw[1] = data[1];
		Cw[2] = data[2];
		Cw[3] = data[0] + data[1] + data[2];
		Cw[4] = data[3];
		Cw[5] = data[4];
		Cw[6] = data[5];
		Cw[7] = data[3] + data[4] + data[5];

		*ParRoundNr = *ParRoundNr + 1;

		//copy cw end
		for(i=0; i<6; i++)
		{
			ParCw6End[i] = data[i];
		}

		if(1)//		if((RoundNr+1) < (BLOCKS_RB * THREADS_RB))
		{
			uchar ksfull[7*8];
			KeySchedule07_Xor(ksfull, Cw);

			//plain input
			for(i=0; i<8; i++)
			{
				data[i] = 0;
			}

			//Enc
			for(i=0; i<23; i++)
			{
				block_encrypt_dataOnly_ulong(ksfull, data);
			}
			
			for(i=0; i<8; i++)
			{
				ParCrypt[i] = data[i];
			}
		}
	}
}

__global__ void kernel_csa_rainbow_a(uchar *Cw6EndArray_d,
	const uchar *Cw6StartArray_d, ulong StartRoundNr, DEBUG_STRUCT *Debug_d)
{
	uchar Cw[8];
	//ulong dwCw;
	int Offset;
	Offset = (blockIdx.x * THREADS_RB) + threadIdx.x;

	// optimize by copy const to shared (for all treads)
	// till sync treads

	COPY_CONST_TO_SHARED

	/** sync */
	__syncthreads();

	/////////////////////////////////
	//6 -> 8
	const uchar *Cw6;
	Cw6 = &Cw6StartArray_d[Offset * 6];
	Cw[0] = Cw6[0];
	Cw[1] = Cw6[1];
	Cw[2] = Cw6[2];
	Cw[3] = Cw6[0] + Cw6[1] + Cw6[2];
	Cw[4] = Cw6[3];
	Cw[5] = Cw6[4];
	Cw[6] = Cw6[5];
	Cw[7] = Cw6[3] + Cw6[4] + Cw6[5];


	int i;
//	int k;
//	int j;
	int RoundNr;

	ulong data[8];

	for(RoundNr=StartRoundNr; RoundNr < (StartRoundNr + 
		RAINBOW_CHAIN_LEN_PER_KERNEL); RoundNr++)
	{
		uchar ksfull[7*8];
		KeySchedule07_Xor(ksfull, Cw);

		//plain input
		for(i=0; i<8; i++)
		{
			data[i] = 0;
		}

		//Enc
		for(i=0; i<23; i++)
		{
			block_encrypt_dataOnly_ulong(ksfull, data);
		}

		//Reduction part 2 (XOR RoundNr)
		data[2] ^= ((RoundNr>>24)&0xFF);
		data[3] ^= ((RoundNr>>16)&0xFF);
		data[4] ^= ((RoundNr>>8)&0xFF);
		data[5] ^= (RoundNr&0xFF);

		//checksum
		Cw[0] = data[0];
		Cw[1] = data[1];
		Cw[2] = data[2];
		Cw[3] = data[0] + data[1] + data[2];
		Cw[4] = data[3];
		Cw[5] = data[4];
		Cw[6] = data[5];
		Cw[7] = data[3] + data[4] + data[5];
	}


	for(i=0; i<6; i++)
	{
		Cw6EndArray_d[(Offset * 6) + i] = data[i];
	}

}

/** function exposed to the world ------------------------------------------ */

int CsaCudaRainbowDoXRounds(uchar *Cw6Array_h, ulong *CurRoundNrArray_h,
	ulong *RemRoundsArray_h, DEBUG_STRUCT *Debug_h)
{
	//Cw6Array_h [in/out]
	//CurRoundNrArray_h [in/out]
	//RemRoundsArray_h [in]
	//return 0=ok
	uchar *Cw6Array_d;
	ulong *CurRoundNrArray_d;
	ulong *RemRoundsArray_d;
	DEBUG_STRUCT *Debug_d;
	hipError_t ce;

	//alloc device memory
	ce = hipMalloc((void **) &Cw6Array_d, BLOCKS_RB * THREADS_RB * 6);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &CurRoundNrArray_d, BLOCKS_RB * THREADS_RB *
			sizeof(ulong));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &RemRoundsArray_d, BLOCKS_RB * THREADS_RB * 
			sizeof(ulong));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &Debug_d, sizeof(DEBUG_STRUCT));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Cw6Array
	ce = hipMemcpy(Cw6Array_d, Cw6Array_h, BLOCKS_RB * THREADS_RB * 6, 
			hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input CurRoundNrArray
	ce = hipMemcpy(CurRoundNrArray_d,CurRoundNrArray_h,BLOCKS_RB*THREADS_RB*
			sizeof(ulong), hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input RemRoundsArray
	ce = hipMemcpy(RemRoundsArray_d, RemRoundsArray_h,BLOCKS_RB * THREADS_RB *
			sizeof(ulong), hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Debug
	ce = hipMemcpy(Debug_d, Debug_h, sizeof(DEBUG_STRUCT), hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	/////////////////
	//Start kernel
	kernel_csa_rainbow_do_x_rounds<<<BLOCKS_RB,THREADS_RB,0,0>>>
		(Cw6Array_d, CurRoundNrArray_d, RemRoundsArray_d, Debug_d);
	/////////////////

	//Sync
	while(hipStreamQuery(0)==hipErrorNotReady)
	{
	  int64_t num_cycles = get_cycles(5.0f);
	  //keep CPU utilization near 0% (thanks to old-man)
	  sleep<<< BLOCKS_RB, THREADS_RB, 0, 0 >>>(num_cycles);
	}

	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Cw6Array
	ce = hipMemcpy(Cw6Array_h, Cw6Array_d, BLOCKS_RB * THREADS_RB * 6,
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output CurRoundNrArray
	ce = hipMemcpy(CurRoundNrArray_h, CurRoundNrArray_d, BLOCKS_RB *
			THREADS_RB * sizeof(ulong), hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output RemRoundsArray
	ce = hipMemcpy(RemRoundsArray_h, RemRoundsArray_d, BLOCKS_RB *THREADS_RB*
			sizeof(ulong), hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Debug
	ce = hipMemcpy(Debug_h, Debug_d, sizeof(DEBUG_STRUCT), 
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Sync
	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//free device memory
	ce = hipFree(Cw6Array_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(CurRoundNrArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(RemRoundsArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(Debug_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	return 0;//ok
}

int CsaCudaRainbowFindA(uchar *Cw6EndArray_h, ulong *RoundNrArray_h,
	uchar *CryptArray_h, DEBUG_STRUCT *Debug_h)
{
	//return 0=ok
	uchar *Cw6EndArray_d;
	uchar *CryptArray_d;
	ulong *RoundNrArray_d;
	DEBUG_STRUCT *Debug_d;
	hipError_t ce;

	//alloc device memory
	ce = hipMalloc((void **) &Cw6EndArray_d, BLOCKS_RB * THREADS_RB * 6);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &RoundNrArray_d, BLOCKS_RB * THREADS_RB * 
			sizeof(ulong));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &CryptArray_d, BLOCKS_RB * THREADS_RB * 8);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &Debug_d, sizeof(DEBUG_STRUCT));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Cw6EndArray
	ce = hipMemcpy(Cw6EndArray_d, Cw6EndArray_h, BLOCKS_RB * THREADS_RB * 6, 
			hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input RoundNrArray
	ce = hipMemcpy(RoundNrArray_d, RoundNrArray_h, BLOCKS_RB * THREADS_RB * 
			sizeof(ulong), hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input CryptArray
	ce = hipMemcpy(CryptArray_d, CryptArray_h, BLOCKS_RB * THREADS_RB * 8,
			hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Debug
	ce = hipMemcpy(Debug_d, Debug_h, sizeof(DEBUG_STRUCT), hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	/////////////////
	//Start kernel
	kernel_csa_rainbow_find_a<<<BLOCKS_RB,THREADS_RB,0,0>>>
		(Cw6EndArray_d,RoundNrArray_d, CryptArray_d, Debug_d);
	/////////////////

	//Sync
	while(hipStreamQuery(0)==hipErrorNotReady)
	{
	  int64_t num_cycles = get_cycles(5.0f);
	  //keep CPU utilization near 0% (thanks to old-man)
	  sleep<<< BLOCKS_RB, THREADS_RB, 0, 0 >>>(num_cycles);
	}

	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Cw6EndArray
	ce = hipMemcpy(Cw6EndArray_h,Cw6EndArray_d,BLOCKS_RB * THREADS_RB * 6,
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output RoundNrArray
	ce = hipMemcpy(RoundNrArray_h, RoundNrArray_d, BLOCKS_RB * THREADS_RB * 
			sizeof(ulong), hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output CryptArray
	ce = hipMemcpy(CryptArray_h, CryptArray_d, BLOCKS_RB * THREADS_RB * 8, 
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Debug
	ce = hipMemcpy(Debug_h, Debug_d, sizeof(DEBUG_STRUCT), 
		hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Sync
	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//free device memory
	ce = hipFree(Cw6EndArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(RoundNrArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(CryptArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(Debug_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	return 0;//ok
}

int CsaCudaRainbow1(uchar *Cw6EndArray_h, const uchar *Cw6StartArray_h,
	ulong StartRoundNr, DEBUG_STRUCT *Debug_h)
{
	uchar *Cw6EndArray_d;
	uchar *Cw6StartArray_d;
	//ulong StartRoundNr_d;
	DEBUG_STRUCT *Debug_d;
	hipError_t ce;

//int deviceCount = 0;
//hipError_t err = hipGetDeviceCount(&deviceCount);
//CheckConditionXR_(err == hipSuccess, err);
	
	//alloc device memory
	ce = hipMalloc((void **) &Cw6StartArray_d, BLOCKS_RB * THREADS_RB * 6);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &Cw6EndArray_d, BLOCKS_RB * THREADS_RB * 6);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipMalloc((void **) &Debug_d, sizeof(DEBUG_STRUCT));
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Cw6StartArray
	ce = hipMemcpy(Cw6StartArray_d, Cw6StartArray_h,BLOCKS_RB*THREADS_RB*6,
			hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy input Debug
	ce = hipMemcpy(Debug_d, Debug_h, sizeof(DEBUG_STRUCT),
			hipMemcpyHostToDevice);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	/////////////////
	//Start kernel
	kernel_csa_rainbow_a<<<BLOCKS_RB,THREADS_RB,0,0>>> 
		( Cw6EndArray_d,Cw6StartArray_d, StartRoundNr, Debug_d);
	/////////////////

	//Sync
	while(hipStreamQuery(0)==hipErrorNotReady)
	{
	  int64_t num_cycles = get_cycles(5.0f);
	  //keep CPU utilization near 0% (thanks to old-man)
	  sleep<<< BLOCKS_RB, THREADS_RB, 0, 0 >>>(num_cycles);
	}

	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Cw6EndArray
	ce = hipMemcpy(Cw6EndArray_h, Cw6EndArray_d, BLOCKS_RB * THREADS_RB * 6,
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Copy output Debug
	ce = hipMemcpy(Debug_h, Debug_d, sizeof(DEBUG_STRUCT),
			hipMemcpyDeviceToHost);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//Sync
	ce = hipStreamSynchronize(0);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	//free device memory
	ce = hipFree(Cw6StartArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(Cw6EndArray_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	ce = hipFree(Debug_d);
	assert(ce == hipSuccess);
	if(ce != hipSuccess){return -1;}

	return 0;//ok
}

